
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void backet_sort(int *key, int *bucket, int *offset, int n, int range, int*b)
{
  int i = threadIdx.x;
  bucket[i] = 0;
  offset[i] = 0;
  __syncthreads();

  if(i==0)
  {
    for (int j=0; j<n; j++)
      bucket[key[j]]++;
  }
  __syncthreads();

  // scan
  if (i!=0) offset[i] = bucket[i-1];
  for (int j=1; j<range; j<<=1)
  {
    b[i] = offset[i];
    __syncthreads();
    if(i>=j) offset[i] += b[i-j];
    __syncthreads();
  }

  for (int j=0; bucket[i]>0; bucket[i]--, ++j) {
    key[offset[i]+j] = i;
  }
}


int main() {
  int n = 50;
  int range = 5;
  int *key, *bucket, *offset, *b ;
  hipMallocManaged( &key, n*sizeof(int) );
  hipMallocManaged( &bucket, range*sizeof(int) );
  hipMallocManaged( &offset, range*sizeof(int) );
  hipMallocManaged( &b, range*sizeof(int) );

  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  backet_sort<<<1, range>>>(key, bucket, offset, n, range, b);
  hipDeviceSynchronize();
  
  hipFree(b);
  hipFree(offset);
  hipFree(bucket);

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(key);
}
